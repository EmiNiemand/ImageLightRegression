#include "hip/hip_runtime.h"
#include "NeuralNetwork/CUDAFunctions.cuh"

#include "NeuralNetwork/AdamOptimizer.h"

#pragma region CUDA
__global__ void CUDAConvLayer(const float* input, float* output, const float* kernel, const float* biases,
                              int inputDimX, int inputDimY, int outputDimX, int outputDimY, int kernelDimX,
                              int kernelDimY, int kernelDimZ, int strideDimX, int strideDimY, int paddingDimX,
                              int paddingDimY, int kernelNumber) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < inputDimX * inputDimY * kernelDimX * kernelDimY) {
        unsigned int inputIdx = idx % (inputDimX * inputDimY);
        unsigned int x = inputIdx % inputDimX;
        unsigned int y = inputIdx / inputDimX;
        unsigned int kx = (idx / (inputDimX * inputDimY)) % kernelDimX;
        unsigned int ky = idx / ((inputDimX * inputDimY) * kernelDimX);

        unsigned int outputIdx = inputIdx + kernelNumber * outputDimX * outputDimY;

        for (int kz = 0; kz < kernelDimZ; ++kz) {
            int index = x * strideDimX - paddingDimX + kx + (y * strideDimY - paddingDimY + ky) * inputDimX;

            if (index < 0 || index >= inputDimX * inputDimY) {
                output[outputIdx] += 0;
            }
            else {
                output[outputIdx] += input[inputIdx+ kz * inputDimX * inputDimY] *
                        kernel[kx + ky * kernelDimX + kz * kernelDimX * kernelDimY];
                if (biases != nullptr) {
                    output[outputIdx] += biases[kernelNumber];
                }
            }
        }
    }
}

__global__ void CUDAReLULayer(float* input, int size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        if (input[idx] < 0) {
            input[idx] = 0;
        }
    }
}

__global__ void CUDAPoolingLayer(const float* input, float* output, int outputDimX, int outputDimY, int outputDimZ,
                                 int poolDimX, int poolDimY, int strideDimX, int strideDimY) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < outputDimX * outputDimY * outputDimZ) {
        unsigned int x = idx % outputDimX;
        unsigned int y = (idx / outputDimX) % outputDimY;
        unsigned int z = idx / (outputDimX * outputDimY);

        unsigned int inputWidth = strideDimX * outputDimX;
        unsigned int inputHeight = strideDimY * outputDimY;

        float max = input[x * strideDimX + y * strideDimY * inputWidth + z * inputWidth * inputHeight];

        for (int ky = 0; ky < poolDimY; ++ky) {
            for (int kx = 0; kx < poolDimX; ++kx) {
                unsigned int inputX = x * strideDimX + kx;
                unsigned int inputY = y * strideDimY + ky;
                unsigned int inputZ = z;

                int index = inputX + inputY * inputWidth + inputZ * inputWidth * inputHeight;
                if (input[index] > max) {
                    max = input[index];
                }
            }
        }

        output[idx] = max;
    }
}

__global__ void CUDAFullyConnectedLayer(const float* input, const float* weights, const float* biases,
                                        float* output, int inputSize, int outputSize) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < outputSize) {
        float neuronValue = 0.0f;

        for (int i = 0; i < inputSize; i++) {
            neuronValue += input[i] * weights[i + idx * inputSize];
        }
        output[idx] = neuronValue;

        if (biases != nullptr) {
            output[idx] += biases[idx];
        }
    }
}

__global__ void CUDAConvLayerGradients(float* prevGradients, float* weightGradients, const float* currentGradients,
                                       const float* prevLayer, const float* weights, int prevWidth, int prevHeight,
                                       int prevDepth, int currentWidth, int currentHeight, int currentDepth,
                                       int kernelWidth, int kernelHeight) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < prevWidth * prevHeight * prevDepth) {
        unsigned int x = idx % prevWidth;
        unsigned int y = (idx / prevWidth) % prevHeight;
        unsigned int z = idx / (prevWidth * prevHeight);

        for (int d = 0; d < currentDepth; ++d) {
            int currentIdx = d * currentWidth * currentHeight + y * currentWidth + x;

            for (int kh = 0; kh < kernelHeight; ++kh) {
                for (int kw = 0; kw < kernelWidth; ++kw) {
                    int weightIdx = z * kernelWidth * kernelHeight * currentDepth + d * kernelWidth * kernelHeight +
                                    kh * kernelWidth + kw;

                    // Input Gradients
                    atomicAdd(&prevGradients[idx], currentGradients[currentIdx] * weights[weightIdx]);
                    // Weight Gradients
                    atomicAdd(&weightGradients[weightIdx], currentGradients[currentIdx] * prevLayer[idx]);
                }
            }
        }
    }
}

__global__ void CUDAConvLayerBiasGradients(float* biasGradients, const float* currentGradients, int currentWidth,
                                           int currentHeight, int currentDepth) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < currentWidth * currentHeight * currentDepth) {
        unsigned int z = idx / (currentWidth * currentHeight);
        // Bias Gradients
        biasGradients[z] += currentGradients[idx];
    }
}

__global__ void CUDAClipGradient(float* gradient, int size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        if (gradient[idx] > CLIP_VALUE) {
            gradient[idx] = CLIP_VALUE;
        }
        else if (gradient[idx] < -CLIP_VALUE){
            gradient[idx] = -CLIP_VALUE;
        }
    }
}
#pragma endregion


Layer* ConvolutionLayer(const Layer* currentLayer, const Group* filters, const ivec2 &stride,
                        const ivec2 &padding, const float* biases) {

    int width = (currentLayer->width - filters->filters[0].width + 2 * padding.x) / stride.x + 1;
    int height = (currentLayer->height - filters->filters[0].height + 2 * padding.y) / stride.y + 1;

    int currentLayerSize = currentLayer->width * currentLayer->height * currentLayer->depth;
    int nextLayerSize = width * height * filters->count;

    Layer* nextLayer = new Layer();
    nextLayer->depth = filters->count;
    nextLayer->width = width;
    nextLayer->height = height;
    nextLayer->maps = new float[nextLayerSize];

    int numBytesCurrentLayerSize = (int)(currentLayerSize * sizeof(float));
    int numBytesNextLayerSize = (int)(nextLayerSize * sizeof(float));

    float* deviceCurrentLayer;
    hipMalloc((void**)&deviceCurrentLayer, numBytesCurrentLayerSize);
    hipMemcpy(deviceCurrentLayer, currentLayer->maps, numBytesCurrentLayerSize, hipMemcpyHostToDevice);

    float* deviceNextLayer;
    hipMalloc((void**)&deviceNextLayer, numBytesNextLayerSize);
    hipMemset(deviceNextLayer, 0, numBytesNextLayerSize);

    float* deviceBiases = nullptr;
    if (biases != nullptr) {
        hipMalloc((void**)&deviceBiases, filters->count * sizeof(float));
        hipMemcpy(deviceBiases, biases, filters->count * sizeof(float), hipMemcpyHostToDevice);
    }

    int numBytesKernelSize = (int)(filters->filters[0].width * filters->filters[0].height * filters->filters[0].depth *
            sizeof(float));

    int blockSize = 512;
    int gridSize = (currentLayer->width * currentLayer->height * filters->filters[0].width * filters->filters[0].height +
            blockSize - 1) / blockSize;

    float* deviceKernels;
    hipMalloc((void**)&deviceKernels, numBytesKernelSize);

    for (int i = 0; i < filters->count; ++i) {
        hipMemcpy(deviceKernels, filters->filters[i].maps, numBytesKernelSize, hipMemcpyHostToDevice);

        CUDAConvLayer<<<gridSize, blockSize>>>(deviceCurrentLayer, deviceNextLayer, deviceKernels, deviceBiases,
                                               currentLayer->width, currentLayer->height, nextLayer->width,
                                               nextLayer->height, filters->filters[i].width, filters->filters[i].height,
                                               filters->filters[i].depth, stride.x, stride.y, padding.x, padding.y, i);
    }
    hipFree(deviceKernels);

    hipMemcpy(nextLayer->maps, deviceNextLayer, numBytesNextLayerSize, hipMemcpyDeviceToHost);
    hipFree(deviceCurrentLayer);
    hipFree(deviceNextLayer);

    if (biases != nullptr) {
        hipFree(deviceBiases);
    }

    return nextLayer;
}

Gradient* ConvolutionLayerBackward(Layer *currentLayer, Group *weights, Layer *previousLayer, std::vector<float>& gradient) {
    int currentGradientSize = (int)gradient.size();
    int previousGradientSize = previousLayer->width * previousLayer->height * previousLayer->depth;
    int weightMapSize = weights->filters[0].width * weights->filters[0].height * weights->filters[0].depth;
    int weightSize = weights->count * weightMapSize;

    Gradient* previousGradient = new Gradient();
    previousGradient->inputsGradients.resize(previousGradientSize, 0.0f);
    previousGradient->weightsGradients.resize(weightSize, 0.0f);
    previousGradient->biasesGradients.resize(weights->count);

    std::vector<float> squashedWeights(weightSize, 0.0f);

    for (int i = 0; i < weights->count; ++i) {
        std::memcpy(&squashedWeights[0] + i * weightMapSize, weights->filters[i].maps, weightMapSize * sizeof(float));
    }

    int numBytesCurrentGradientSize = (int)(currentGradientSize * sizeof(float));
    int numBytesPreviousGradientSize = (int)(previousGradientSize * sizeof(float));
    int numBytesWeightsSize = (int)(weightSize * sizeof(float));

    float* deviceGradient;
    hipMalloc((void**)&deviceGradient, numBytesCurrentGradientSize);
    hipMemcpy(deviceGradient, gradient.data(), numBytesCurrentGradientSize, hipMemcpyHostToDevice);

    float* devicePreviousGradient;
    hipMalloc((void**)&devicePreviousGradient, numBytesPreviousGradientSize);
    hipMemset(devicePreviousGradient, 0, numBytesPreviousGradientSize);

    float* deviceWeightGradient;
    hipMalloc((void**)&deviceWeightGradient, numBytesWeightsSize);
    hipMemset(deviceWeightGradient, 0, numBytesWeightsSize);

    float* deviceBiasesGradient;
    hipMalloc((void**)&deviceBiasesGradient, weights->count * sizeof(float));
    hipMemset(deviceBiasesGradient, 0, weights->count * sizeof(float));

    float* devicePreviousLayer;
    hipMalloc((void**)&devicePreviousLayer, numBytesPreviousGradientSize);
    hipMemcpy(devicePreviousLayer, previousLayer->maps, numBytesPreviousGradientSize, hipMemcpyHostToDevice);

    float* deviceWeights;
    hipMalloc((void**)&deviceWeights, numBytesWeightsSize);
    hipMemcpy(deviceWeights, squashedWeights.data(), numBytesWeightsSize, hipMemcpyHostToDevice);

    int blockSize = 512;
    int gridSize = (previousGradientSize + blockSize - 1) / blockSize;

    CUDAConvLayerGradients<<<gridSize, blockSize>>>(devicePreviousGradient, deviceWeightGradient, deviceGradient,
                                                    devicePreviousLayer, deviceWeights, previousLayer->width,
                                                    previousLayer->height, previousLayer->depth, currentLayer->width,
                                                    currentLayer->height, currentLayer->depth,
                                                    weights->filters[0].width,
                                                    weights->filters[0].height);

    gridSize = (currentGradientSize + blockSize - 1) / blockSize;

    CUDAConvLayerBiasGradients<<<gridSize, blockSize>>>(deviceBiasesGradient, deviceGradient, currentLayer->width,
                                                        currentLayer->height, currentLayer->depth);

    hipMemcpy(previousGradient->inputsGradients.data(), devicePreviousGradient, numBytesPreviousGradientSize, hipMemcpyDeviceToHost);
    hipMemcpy(previousGradient->weightsGradients.data(), deviceWeightGradient, numBytesWeightsSize, hipMemcpyDeviceToHost);
    hipMemcpy(previousGradient->biasesGradients.data(), deviceBiasesGradient, weights->count * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceWeightGradient);
    hipFree(deviceGradient);
    hipFree(deviceBiasesGradient);
    hipFree(devicePreviousGradient);
    hipFree(devicePreviousLayer);
    hipFree(deviceWeights);

    ClipGradient(previousGradient->inputsGradients);
    ClipGradient(previousGradient->weightsGradients);
    ClipGradient(previousGradient->biasesGradients);

    return previousGradient;
}

void ReLULayer(Layer* currentLayer) {
    int currentLayerSize = currentLayer->width * currentLayer->height * currentLayer->depth;
    int numBytesCurrentLayerSize = (int)(currentLayerSize * sizeof(float));

    int blockSize = 256;
    int gridSize = (currentLayerSize + blockSize - 1) / blockSize;

    float* deviceCurrentLayer;
    hipMalloc((void**)&deviceCurrentLayer, numBytesCurrentLayerSize);
    hipMemcpy(deviceCurrentLayer, currentLayer->maps, numBytesCurrentLayerSize, hipMemcpyHostToDevice);

    CUDAReLULayer<<<gridSize, blockSize>>>(deviceCurrentLayer, currentLayerSize);

    hipMemcpy(currentLayer->maps, deviceCurrentLayer, numBytesCurrentLayerSize, hipMemcpyDeviceToHost);
    hipFree(deviceCurrentLayer);
}

Layer* PoolingLayer(const Layer* currentLayer, const ivec2& poolDim, const ivec2& stride) {
    int width = (currentLayer->width - poolDim.x) / stride.x + 1;
    int height = (currentLayer->height - poolDim.y) / stride.y + 1;

    int currentLayerSize = currentLayer->width * currentLayer->height * currentLayer->depth;
    int nextLayerSize = width * height * currentLayer->depth;

    Layer* nextLayer = new Layer();
    nextLayer->depth = currentLayer->depth;
    nextLayer->width = width;
    nextLayer->height = height;
    nextLayer->maps = new float[nextLayerSize];

    int blockSize = 256;
    int gridSize = (nextLayerSize + blockSize - 1) / blockSize;

    int numBytesCurrentLayerSize = (int)(currentLayerSize * sizeof(float));
    int numBytesNextLayerSize = (int)(nextLayerSize * sizeof(float));

    float* deviceCurrentLayer;
    hipMalloc((void**)&deviceCurrentLayer, numBytesCurrentLayerSize);
    hipMemcpy(deviceCurrentLayer, currentLayer->maps, numBytesCurrentLayerSize, hipMemcpyHostToDevice);

    float* deviceNextLayer;
    hipMalloc((void**)&deviceNextLayer, numBytesNextLayerSize);

    CUDAPoolingLayer<<<gridSize, blockSize>>>(deviceCurrentLayer, deviceNextLayer, width, height, nextLayer->depth,
                                              poolDim.x, poolDim.y, stride.x, stride.y);

    hipMemcpy(nextLayer->maps, deviceNextLayer, numBytesNextLayerSize, hipMemcpyDeviceToHost);

    hipFree(deviceCurrentLayer);
    hipFree(deviceNextLayer);

    return nextLayer;
}

void MaxPoolingBackward(const Layer* currentLayer, const Layer* previousLayer, std::vector<float>& gradient,
                                      ivec2 poolDim, ivec2 strideDim) {
    std::vector<float> previousGradient(previousLayer->width * previousLayer->height * previousLayer->depth, 0.0f);

    for (int d = 0; d < previousLayer->depth; ++d) {
        for (int h = 0; h < previousLayer->height; h+=strideDim.y) {
            for (int w = 0; w < previousLayer->width; w+=strideDim.x) {
                int currIdx = w / strideDim.x + h / strideDim.y * currentLayer->width +
                        d * currentLayer->width * currentLayer->height;
                bool foundMatch = false;

                for (int poolY = 0; poolY < poolDim.y; ++poolY) {
                    for (int poolX = 0; poolX < poolDim.x; ++poolX) {
                        int prevIdx = w + poolX + (h + poolY) * previousLayer->width +
                                d * previousLayer->width * previousLayer->height;

                        if (previousLayer->maps[prevIdx] == currentLayer->maps[currIdx]) {
                            previousGradient[prevIdx] = gradient[currIdx];
                            foundMatch = true;
                            break;
                        }
                    }

                    if (foundMatch) {
                        break;  // Break out of the poolY loop
                    }
                }
            }
        }
    }

    gradient.clear();
    gradient = previousGradient;
}

Layer* FullyConnectedLayer(const Layer* currentLayer, const float* weights, int currentLayerSize, int nextLayerSize,
                           const float* biases) {
    Layer* nextLayer = new Layer();
    nextLayer->depth = 1;
    nextLayer->height = 1;
    nextLayer->width = nextLayerSize;
    nextLayer->maps = new float[nextLayerSize];

    int numBytesCurrentLayerSize = (int)(currentLayerSize * sizeof(float));
    int numBytesNexLayerSize = (int)(nextLayerSize * sizeof(float));

    // Allocate memory on the device (GPU)
    float* deviceCurrentLayerNeurons;
    float* deviceWeights;
    float* deviceBiases = nullptr;
    float* deviceNextLayerNeurons;
    hipMalloc((void**)&deviceCurrentLayerNeurons, numBytesCurrentLayerSize);
    hipMalloc((void**)&deviceWeights, numBytesCurrentLayerSize * nextLayerSize);
    hipMalloc((void**)&deviceNextLayerNeurons, numBytesNexLayerSize);
    hipMemset((void**)&deviceNextLayerNeurons, 0, numBytesNexLayerSize);

    if (biases != nullptr) {
        hipMalloc((void**)&deviceBiases, numBytesNexLayerSize);
        hipMemcpy(deviceBiases, biases, numBytesNexLayerSize, hipMemcpyHostToDevice);
    }

    // Copy input neurons and weights from host to device
    hipMemcpy(deviceCurrentLayerNeurons, currentLayer->maps, numBytesCurrentLayerSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceWeights, weights, numBytesCurrentLayerSize * nextLayerSize, hipMemcpyHostToDevice);

    int blockSize = 512;
    int gridSize = (currentLayerSize * nextLayerSize + blockSize - 1) / blockSize;

    // Launch the kernel to calculate the neuron values on the GPU
    CUDAFullyConnectedLayer<<<gridSize, blockSize>>>(deviceCurrentLayerNeurons, deviceWeights, deviceBiases, deviceNextLayerNeurons,
                                                     currentLayerSize, nextLayerSize);

    // Copy the result array from device to host
    hipMemcpy(nextLayer->maps, deviceNextLayerNeurons, numBytesNexLayerSize, hipMemcpyDeviceToHost);

    // Free memory on the device
    hipFree(deviceCurrentLayerNeurons);
    hipFree(deviceWeights);
    hipFree(deviceNextLayerNeurons);

    if (deviceBiases != nullptr) {
        hipFree(deviceBiases);
    }

    return nextLayer;
}

Gradient* FullyConnectedLayerBackward(Layer* currentLayer, Group* weights, Layer* previousLayer, std::vector<float>& gradient) {
    Gradient* previousGradient = new Gradient();

    int currentLayerSize = currentLayer->width * currentLayer->height * currentLayer->depth;
    int previousLayerSize = previousLayer->width * previousLayer->height * previousLayer->depth;
    previousGradient->inputsGradients.resize(previousLayerSize, 0.0f);
    previousGradient->weightsGradients.resize(previousLayerSize * currentLayerSize, 0.0f);
    previousGradient->biasesGradients.resize(currentLayerSize, 0.0f);

    for (int i = 0; i < currentLayerSize; ++i) {
        for (int j = 0; j < previousLayerSize; ++j) {
            previousGradient->inputsGradients[j] += gradient[i] * weights->filters[0].maps[j + i * gradient.size()];
            previousGradient->weightsGradients[j + i * currentLayerSize] += gradient[i] * previousLayer->maps[j];
        }
    }

    std::memcpy(previousGradient->biasesGradients.data(), gradient.data(), gradient.size() * sizeof(float));

    ClipGradient(previousGradient->inputsGradients);
    ClipGradient(previousGradient->weightsGradients);
    ClipGradient(previousGradient->biasesGradients);

    return previousGradient;
}


void DropoutLayer(Layer *currentLayer, float probability) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> distribution(0, 1);

    for (int i = 0; i < currentLayer->width * currentLayer->height * currentLayer->depth; ++i) {
        float randomValue = distribution(gen);
        if (randomValue < probability) {
            currentLayer->maps[i] = 0;
        }
        else {
            currentLayer->maps[i] *= 1 / (1 - probability);
        }
    }
}

float MSELossFunction(const float* input, const float* predictedResult, int size) {
    float loss = 0;

    for (int i = 0; i < size; ++i) {
        float diff = input[i] - predictedResult[i];
        loss += diff * diff;
    }

    loss /= (float)size;

    return loss;
}

void ClipGradient(std::vector<float>& gradient) {
    int dataSize = (int)gradient.size();
    int numBytesDataSize = (int)(dataSize * sizeof(float));

    int blockSize = 256;
    int gridSize = (dataSize + blockSize - 1) / blockSize;

    float* deviceData;
    hipMalloc((void**)&deviceData, numBytesDataSize);
    hipMemcpy(deviceData, gradient.data(), numBytesDataSize, hipMemcpyHostToDevice);

    CUDAClipGradient<<<gridSize, blockSize>>>(deviceData, dataSize);

    hipMemcpy(gradient.data(), deviceData, numBytesDataSize, hipMemcpyDeviceToHost);
    hipFree(deviceData);
}


void MiniBatch(const std::vector<std::vector<Gradient*>>& gradients, std::vector<Group*>& weights, std::vector<Layer*>& biases) {
    std::vector<Gradient*> avgGradients;
    avgGradients.reserve(gradients[0].size());

    for (int i = 0; i < gradients[0].size(); ++i) {
        avgGradients.push_back(new Gradient());
        avgGradients[i]->weightsGradients.resize(gradients[0][i]->weightsGradients.size(), 0.0f);
        avgGradients[i]->biasesGradients.resize(gradients[0][i]->biasesGradients.size(), 0.0f);
    }

    for (int gradient = 0; gradient < gradients.size(); ++gradient) {
        for (int i = 0; i < gradients[gradient].size(); ++i) {
            for (int j = 0; j < gradients[gradient][i]->weightsGradients.size(); ++j) {
                avgGradients[i]->weightsGradients[j] += (gradients[gradient][i]->weightsGradients[j] / (float)gradients.size());
            }

            for (int j = 0; j < gradients[gradient][i]->biasesGradients.size(); ++j) {
                avgGradients[i]->biasesGradients[j] += (gradients[gradient][i]->biasesGradients[j] / (float)gradients.size());
            }
        }
    }

    UpdateWeightsAndBiases(avgGradients, weights, biases);

    for (int i = 0; i < avgGradients.size(); ++i) {
        delete avgGradients[i];
    }
}

void UpdateWeightsAndBiases(const std::vector<Gradient*>& gradients, std::vector<Group*>& weights, std::vector<Layer*>& biases) {
    AdamOptimizer* adamOptimizer = AdamOptimizer::GetInstance();

    for (int layer = 0; layer < gradients.size(); ++layer) {
        int idx = 15 - layer;
        adamOptimizer->UpdateParameters(biases[idx]->maps,  biases[idx]->width * biases[idx]->height * biases[idx]->depth,
                                        gradients[layer]->biasesGradients);

        for (int i = 0; i < weights[idx]->count; ++i) {
            int weightsSize = weights[idx]->filters[i].width * weights[idx]->filters[i].height * weights[idx]->filters[i].depth;
            std::vector<float> weightGradients(weightsSize);

            std::memcpy(&weightGradients[0], &gradients[layer]->weightsGradients[0] + i * weightsSize, weightsSize * sizeof(float));

            adamOptimizer->UpdateParameters(weights[idx]->filters[i].maps, weightsSize, weightGradients);
        }
    }
}
