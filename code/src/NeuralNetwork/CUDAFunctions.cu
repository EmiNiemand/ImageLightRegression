#include "hip/hip_runtime.h"
#include "NeuralNetwork/CUDAFunctions.cuh"

#include "NeuralNetwork/AdamOptimizer.h"

#pragma region CUDA
__global__ void CUDAConvLayer(const float* input, float* output, const float* kernel, const float* biases,
                              int inputDimX, int inputDimY, int outputDimX, int outputDimY, int kernelDimX,
                              int kernelDimY, int kernelDimZ, int strideDimX, int strideDimY, int paddingDimX,
                              int paddingDimY, int kernelNumber) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < outputDimX * outputDimY) {
        unsigned int x = idx % outputDimX;
        unsigned int y = idx / outputDimX;

        unsigned int outputIdx = idx + kernelNumber * outputDimX * outputDimY;

        for (int kz = 0; kz < kernelDimZ; ++kz) {
            for (int ky = 0; ky < kernelDimY; ++ky) {
                for (int kx = 0; kx < kernelDimX; ++kx) {
                    int inputX = kx + x * strideDimX - paddingDimX;
                    int inputY = ky + y * strideDimY - paddingDimY;

                    if (inputX >= 0 && inputX < inputDimX && inputY >= 0 && inputY < inputDimY) {
                        int inputIdx = inputX + inputY * inputDimX + kz * inputDimX * inputDimY;
                        int kernelIdx = kx + ky * kernelDimX + kz * kernelDimX * kernelDimY;

                        output[outputIdx] += input[inputIdx] * kernel[kernelIdx];
                    }
                }
            }
        }

        if (biases != nullptr) {
            output[outputIdx] += biases[kernelNumber];
        }
    }
}

__global__ void CUDAReLULayer(float* input, int size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        if (input[idx] < 0) {
            input[idx] = 0;
        }
    }
}

__global__ void CUDAMaxPoolingLayer(const float* input, float* output, int outputDimX, int outputDimY, int outputDimZ,
                                    int poolDimX, int poolDimY, int strideDimX, int strideDimY) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < outputDimX * outputDimY * outputDimZ) {
        unsigned int x = idx % outputDimX;
        unsigned int y = (idx / outputDimX) % outputDimY;
        unsigned int z = idx / (outputDimX * outputDimY);

        unsigned int inputWidth = strideDimX * outputDimX;
        unsigned int inputHeight = strideDimY * outputDimY;

        float max = input[x * strideDimX + y * strideDimY * inputWidth + z * inputWidth * inputHeight];

        for (int ky = 0; ky < poolDimY; ++ky) {
            for (int kx = 0; kx < poolDimX; ++kx) {
                unsigned int inputX = kx + x * strideDimX;
                unsigned int inputY = ky + y * strideDimY;
                unsigned int inputZ = z;

                int index = inputX + inputY * inputWidth + inputZ * inputWidth * inputHeight;
                if (input[index] > max) {
                    max = input[index];
                }
            }
        }

        output[idx] = max;
    }
}

__global__ void CUDAFullyConnectedLayer(const float* input, const float* weights, const float* biases,
                                        float* output, int inputSize, int outputSize) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < outputSize) {
        float neuronValue = 0.0f;

        for (int i = 0; i < inputSize; i++) {
            neuronValue += input[i] * weights[i + idx * inputSize];
        }

        if (biases != nullptr) {
            neuronValue += biases[idx];
        }

        output[idx] = neuronValue;
    }
}

__global__ void CUDAConvLayerBackward(float* prevGradients, float* weightGradients, float* biasGradients,
                                      const float* currentGradients, const float* prevLayer,
                                      const float* weights, int prevWidth, int prevHeight, int prevDepth,
                                      int currentWidth, int currentHeight, int currentDepth,
                                      int kernelWidth, int kernelHeight) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < prevWidth * prevHeight * prevDepth) {
        unsigned int x = idx % prevWidth;
        unsigned int y = (idx / prevWidth) % prevHeight;
        unsigned int z = idx / (prevWidth * prevHeight);

        for (int kh = 0; kh < kernelHeight; ++kh) {
            for (int kw = 0; kw < kernelWidth; ++kw) {
                for (int d = 0; d < currentDepth; ++d) {
                    int currentIdx = x + y * currentWidth + d * currentWidth * currentHeight;
                    int weightIdx = kw + kh * kernelWidth + d * kernelWidth * kernelHeight +
                            z * kernelWidth * kernelHeight * currentDepth;

                    // Input Gradients
                    atomicAdd(&prevGradients[idx], currentGradients[currentIdx] * weights[weightIdx]);
                    // Weight Gradients
                    atomicAdd(&weightGradients[weightIdx], currentGradients[currentIdx] * prevLayer[idx]);
                }
            }
        }

        // Bias Gradients
        atomicAdd(&biasGradients[z], currentGradients[idx]);
    }
}

__global__ void CUDAMaxPoolingLayerBackward(float* prevGradients, const float* currentGradients, const float* prevLayer,
                                            int currWidth, int currHeight, int currDepth, int prevWidth, int prevHeight,
                                            int poolDimX, int poolDimY, int strideDimX, int strideDimY) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < currWidth * currHeight * currDepth) {
        unsigned int x = idx % currWidth;
        unsigned int y = (idx / currWidth) % currHeight;
        unsigned int z = idx / (currWidth * currHeight);

        int maxPos = -1;
        float maxValue = -FLT_MAX;

        for (int poolY = 0; poolY < poolDimY; ++poolY) {
            for (int poolX = 0; poolX < poolDimX; ++poolX) {
                int inputX = poolX + x * strideDimX;
                int inputY = poolY + y * strideDimY;

                int inputIdx = inputX + inputY * prevWidth + z * prevWidth * prevHeight;

                if (prevLayer[inputIdx] > maxValue) {
                    maxValue = prevLayer[inputIdx];
                    maxPos = inputIdx;
                }
            }
        }

        atomicAdd(&prevGradients[maxPos], currentGradients[idx]);
    }
}

__global__ void CUDAFullyConnectedLayerBackward(float* prevGradients, float* weightGradients, const float* prevLayer,
                                                const float* weights, const float* currentGradients, int prevSize, int currentSize) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < prevSize * currentSize) {
        unsigned int x = idx % prevSize;
        unsigned int y = idx / prevSize;

        weightGradients[idx] += currentGradients[y] * prevLayer[x];
        atomicAdd(&prevGradients[x], currentGradients[y] * weights[idx]);
    }
}

__global__ void CUDAClipGradient(float* gradient, int size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        if (gradient[idx] > CLIP_VALUE) {
            gradient[idx] = CLIP_VALUE;
        }
        else if (gradient[idx] < -CLIP_VALUE){
            gradient[idx] = -CLIP_VALUE;
        }
    }
}
#pragma endregion


Layer* ConvolutionLayer(const Layer* currentLayer, const Group* filters, const ivec2 &stride,
                        const ivec2 &padding, const float* biases) {

    int width = (currentLayer->width - filters->filters[0].width + 2 * padding.x) / stride.x + 1;
    int height = (currentLayer->height - filters->filters[0].height + 2 * padding.y) / stride.y + 1;

    int currentLayerSize = currentLayer->width * currentLayer->height * currentLayer->depth;
    int nextLayerSize = width * height * filters->count;

    Layer* nextLayer = new Layer();
    nextLayer->depth = filters->count;
    nextLayer->width = width;
    nextLayer->height = height;
    nextLayer->maps = new float[nextLayerSize];

    int numBytesCurrentLayerSize = (int)(currentLayerSize * sizeof(float));
    int numBytesNextLayerSize = (int)(nextLayerSize * sizeof(float));

    float* deviceCurrentLayer;
    hipMalloc((void**)&deviceCurrentLayer, numBytesCurrentLayerSize);
    hipMemcpy(deviceCurrentLayer, currentLayer->maps, numBytesCurrentLayerSize, hipMemcpyHostToDevice);

    float* deviceNextLayer;
    hipMalloc((void**)&deviceNextLayer, numBytesNextLayerSize);
    hipMemset(deviceNextLayer, 0, numBytesNextLayerSize);

    float* deviceBiases = nullptr;
    if (biases != nullptr) {
        hipMalloc((void**)&deviceBiases, filters->count * sizeof(float));
        hipMemcpy(deviceBiases, biases, filters->count * sizeof(float), hipMemcpyHostToDevice);
    }

    int numBytesKernelSize = (int)(filters->filters[0].width * filters->filters[0].height * filters->filters[0].depth *
                                   sizeof(float));

    float* deviceKernels;
    hipMalloc((void**)&deviceKernels, numBytesKernelSize);

    int blockSize = 512;
    int gridSize = (currentLayer->width * currentLayer->height + blockSize - 1) / blockSize;

    for (int i = 0; i < filters->count; ++i) {
        hipMemcpy(deviceKernels, filters->filters[i].maps, numBytesKernelSize, hipMemcpyHostToDevice);

        CUDAConvLayer<<<gridSize, blockSize>>>(deviceCurrentLayer, deviceNextLayer, deviceKernels, deviceBiases,
                                               currentLayer->width, currentLayer->height, nextLayer->width,
                                               nextLayer->height, filters->filters[i].width, filters->filters[i].height,
                                               filters->filters[i].depth, stride.x, stride.y, padding.x, padding.y, i);
    }
    hipMemcpy(nextLayer->maps, deviceNextLayer, numBytesNextLayerSize, hipMemcpyDeviceToHost);
    hipFree(deviceKernels);
    hipFree(deviceCurrentLayer);
    hipFree(deviceNextLayer);

    if (biases != nullptr) {
        hipFree(deviceBiases);
    }

    return nextLayer;
}

Gradient* ConvolutionLayerBackward(Layer *currentLayer, Group *weights, Layer *previousLayer, std::vector<float>& gradient) {
    int currentGradientSize = (int)gradient.size();
    int previousGradientSize = previousLayer->width * previousLayer->height * previousLayer->depth;
    int weightMapSize = weights->filters[0].width * weights->filters[0].height * weights->filters[0].depth;
    int weightSize = weights->count * weightMapSize;

    Gradient* previousGradient = new Gradient();
    previousGradient->inputGradients.resize(previousGradientSize, 0.0f);
    previousGradient->weightGradients.resize(weightSize, 0.0f);
    previousGradient->biasGradients.resize(weights->count);

    std::vector<float> squashedWeights(weightSize, 0.0f);

    for (int i = 0; i < weights->count; ++i) {
        std::memcpy(&squashedWeights[0] + i * weightMapSize, weights->filters[i].maps, weightMapSize * sizeof(float));
    }

    int numBytesCurrentGradientSize = (int)(currentGradientSize * sizeof(float));
    int numBytesPreviousGradientSize = (int)(previousGradientSize * sizeof(float));
    int numBytesWeightsSize = (int)(weightSize * sizeof(float));

    float* deviceCurrentGradients;
    hipMalloc((void**)&deviceCurrentGradients, numBytesCurrentGradientSize);
    hipMemcpy(deviceCurrentGradients, gradient.data(), numBytesCurrentGradientSize, hipMemcpyHostToDevice);

    float* devicePreviousGradient;
    hipMalloc((void**)&devicePreviousGradient, numBytesPreviousGradientSize);
    hipMemset(devicePreviousGradient, 0, numBytesPreviousGradientSize);

    float* deviceWeightGradients;
    hipMalloc((void**)&deviceWeightGradients, numBytesWeightsSize);
    hipMemset(deviceWeightGradients, 0, numBytesWeightsSize);

    float* deviceBiasGradients;
    hipMalloc((void**)&deviceBiasGradients, weights->count * sizeof(float));
    hipMemset(deviceBiasGradients, 0, weights->count * sizeof(float));

    float* devicePreviousLayer;
    hipMalloc((void**)&devicePreviousLayer, numBytesPreviousGradientSize);
    hipMemcpy(devicePreviousLayer, previousLayer->maps, numBytesPreviousGradientSize, hipMemcpyHostToDevice);

    float* deviceWeights;
    hipMalloc((void**)&deviceWeights, numBytesWeightsSize);
    hipMemcpy(deviceWeights, squashedWeights.data(), numBytesWeightsSize, hipMemcpyHostToDevice);

    int blockSize = 512;
    int gridSize = (previousGradientSize + blockSize - 1) / blockSize;

    CUDAConvLayerBackward<<<gridSize, blockSize>>>(devicePreviousGradient, deviceWeightGradients, deviceBiasGradients,
                                                   deviceCurrentGradients, devicePreviousLayer, deviceWeights,
                                                   previousLayer->width, previousLayer->height, previousLayer->depth,
                                                   currentLayer->width, currentLayer->height, currentLayer->depth,
                                                   weights->filters[0].width, weights->filters[0].height);


    hipMemcpy(previousGradient->inputGradients.data(), devicePreviousGradient, numBytesPreviousGradientSize, hipMemcpyDeviceToHost);
    hipMemcpy(previousGradient->weightGradients.data(), deviceWeightGradients, numBytesWeightsSize, hipMemcpyDeviceToHost);
    hipMemcpy(previousGradient->biasGradients.data(), deviceBiasGradients, weights->count * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceWeightGradients);
    hipFree(deviceCurrentGradients);
    hipFree(deviceBiasGradients);
    hipFree(devicePreviousGradient);
    hipFree(devicePreviousLayer);
    hipFree(deviceWeights);

    ClipGradient(previousGradient->inputGradients);
    ClipGradient(previousGradient->weightGradients);
    ClipGradient(previousGradient->biasGradients);

    return previousGradient;
}

void ReLULayer(Layer* currentLayer) {
    int currentLayerSize = currentLayer->width * currentLayer->height * currentLayer->depth;
    int numBytesCurrentLayerSize = (int)(currentLayerSize * sizeof(float));

    float* deviceCurrentLayer;
    hipMalloc((void**)&deviceCurrentLayer, numBytesCurrentLayerSize);
    hipMemcpy(deviceCurrentLayer, currentLayer->maps, numBytesCurrentLayerSize, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (currentLayerSize + blockSize - 1) / blockSize;

    CUDAReLULayer<<<gridSize, blockSize>>>(deviceCurrentLayer, currentLayerSize);

    hipMemcpy(currentLayer->maps, deviceCurrentLayer, numBytesCurrentLayerSize, hipMemcpyDeviceToHost);
    hipFree(deviceCurrentLayer);
}

Layer* MaxPoolingLayer(const Layer* currentLayer, const ivec2& poolDim, const ivec2& stride) {
    int width = (currentLayer->width - poolDim.x) / stride.x + 1;
    int height = (currentLayer->height - poolDim.y) / stride.y + 1;

    int currentLayerSize = currentLayer->width * currentLayer->height * currentLayer->depth;
    int nextLayerSize = width * height * currentLayer->depth;

    Layer* nextLayer = new Layer();
    nextLayer->depth = currentLayer->depth;
    nextLayer->width = width;
    nextLayer->height = height;
    nextLayer->maps = new float[nextLayerSize];

    int numBytesCurrentLayerSize = (int)(currentLayerSize * sizeof(float));
    int numBytesNextLayerSize = (int)(nextLayerSize * sizeof(float));

    float* deviceCurrentLayer;
    hipMalloc((void**)&deviceCurrentLayer, numBytesCurrentLayerSize);
    hipMemcpy(deviceCurrentLayer, currentLayer->maps, numBytesCurrentLayerSize, hipMemcpyHostToDevice);

    float* deviceNextLayer;
    hipMalloc((void**)&deviceNextLayer, numBytesNextLayerSize);

    int blockSize = 256;
    int gridSize = (nextLayerSize + blockSize - 1) / blockSize;

    CUDAMaxPoolingLayer<<<gridSize, blockSize>>>(deviceCurrentLayer, deviceNextLayer, width, height, nextLayer->depth,
                                                 poolDim.x, poolDim.y, stride.x, stride.y);

    hipMemcpy(nextLayer->maps, deviceNextLayer, numBytesNextLayerSize, hipMemcpyDeviceToHost);

    hipFree(deviceCurrentLayer);
    hipFree(deviceNextLayer);

    return nextLayer;
}

void MaxPoolingBackward(const Layer* currentLayer, const Layer* previousLayer, std::vector<float>& gradient,
                                      ivec2 poolDim, ivec2 strideDim) {
    int prevLayerSize = previousLayer->width * previousLayer->height * previousLayer->depth;
    int numBytesPrevLayerSize = prevLayerSize * sizeof(float);

    float* devicePrevGradients;
    hipMalloc((void**)&devicePrevGradients, numBytesPrevLayerSize);
    hipMemset(devicePrevGradients, 0, numBytesPrevLayerSize);

    float* deviceGradients;
    hipMalloc((void**)&deviceGradients, gradient.size() * sizeof(float));
    hipMemcpy(deviceGradients, gradient.data(), gradient.size() * sizeof(float), hipMemcpyHostToDevice);

    float* devicePrevLayer;
    hipMalloc((void**)&devicePrevLayer, numBytesPrevLayerSize);
    hipMemcpy(devicePrevLayer, previousLayer->maps, numBytesPrevLayerSize, hipMemcpyHostToDevice);

    int blockSize = 512;
    int gridSize = (currentLayer->width * currentLayer->height * currentLayer->depth + blockSize - 1) / blockSize;

    CUDAMaxPoolingLayerBackward<<<gridSize, blockSize>>>(devicePrevGradients, deviceGradients, devicePrevLayer,
                                                         currentLayer->width, currentLayer->height, currentLayer->depth,
                                                         previousLayer->width, previousLayer->height, poolDim.x,
                                                         poolDim.y, strideDim.x, strideDim.y);

    gradient.clear();
    gradient.resize(prevLayerSize, 0.0f);
    hipMemcpy(gradient.data(), devicePrevGradients, numBytesPrevLayerSize, hipMemcpyDeviceToHost);

    hipFree(devicePrevGradients);
    hipFree(deviceGradients);
    hipFree(devicePrevLayer);
}

Layer* FullyConnectedLayer(const Layer* currentLayer, const float* weights, int currentLayerSize, int nextLayerSize,
                           const float* biases) {
    Layer* nextLayer = new Layer();
    nextLayer->depth = 1;
    nextLayer->height = 1;
    nextLayer->width = nextLayerSize;
    nextLayer->maps = new float[nextLayerSize];

    int numBytesCurrentLayerSize = (int)(currentLayerSize * sizeof(float));
    int numBytesNexLayerSize = (int)(nextLayerSize * sizeof(float));

    float* deviceCurrentLayerNeurons;
    hipMalloc((void**)&deviceCurrentLayerNeurons, numBytesCurrentLayerSize);

    float* deviceWeights;
    hipMalloc((void**)&deviceWeights, numBytesCurrentLayerSize * nextLayerSize);

    float* deviceNextLayerNeurons;
    hipMalloc((void**)&deviceNextLayerNeurons, numBytesNexLayerSize);
    hipMemset((void**)&deviceNextLayerNeurons, 0, numBytesNexLayerSize);

    float* deviceBiases = nullptr;
    if (biases != nullptr) {
        hipMalloc((void**)&deviceBiases, numBytesNexLayerSize);
        hipMemcpy(deviceBiases, biases, numBytesNexLayerSize, hipMemcpyHostToDevice);
    }

    hipMemcpy(deviceCurrentLayerNeurons, currentLayer->maps, numBytesCurrentLayerSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceWeights, weights, numBytesCurrentLayerSize * nextLayerSize, hipMemcpyHostToDevice);

    int blockSize = 512;
    int gridSize = (currentLayerSize * nextLayerSize + blockSize - 1) / blockSize;

    CUDAFullyConnectedLayer<<<gridSize, blockSize>>>(deviceCurrentLayerNeurons, deviceWeights, deviceBiases,
                                                     deviceNextLayerNeurons, currentLayerSize, nextLayerSize);

    hipMemcpy(nextLayer->maps, deviceNextLayerNeurons, numBytesNexLayerSize, hipMemcpyDeviceToHost);

    hipFree(deviceCurrentLayerNeurons);
    hipFree(deviceWeights);
    hipFree(deviceNextLayerNeurons);

    if (deviceBiases != nullptr) {
        hipFree(deviceBiases);
    }

    return nextLayer;
}

Gradient* FullyConnectedLayerBackward(Layer* currentLayer, Group* weights, Layer* previousLayer, std::vector<float>& gradient) {
    Gradient* previousGradient = new Gradient();

    int currentLayerSize = currentLayer->width * currentLayer->height * currentLayer->depth;
    int previousLayerSize = previousLayer->width * previousLayer->height * previousLayer->depth;
    previousGradient->inputGradients.resize(previousLayerSize, 0.0f);
    previousGradient->weightGradients.resize(previousLayerSize * currentLayerSize, 0.0f);
    previousGradient->biasGradients.resize(currentLayerSize, 0.0f);

    int numBytesWeightGradients = (int)(previousLayerSize * currentLayerSize * sizeof(float));
    int numBytesPreviousLayer = (int)(previousLayerSize * sizeof(float));
    int numBytesGradient = (int)(currentLayerSize * sizeof(float));

    float* devicePrevGradients;
    hipMalloc((void**)&devicePrevGradients, numBytesPreviousLayer);
    hipMemset(devicePrevGradients, 0, numBytesPreviousLayer);

    float* deviceWeightGradients;
    hipMalloc((void**)&deviceWeightGradients, numBytesWeightGradients);
    hipMemset(deviceWeightGradients, 0, numBytesWeightGradients);

    float* devicePreviousLayer;
    hipMalloc((void**)&devicePreviousLayer, numBytesPreviousLayer);
    hipMemcpy(devicePreviousLayer, previousLayer->maps, numBytesPreviousLayer, hipMemcpyHostToDevice);

    float* deviceWeights;
    hipMalloc((void**)&deviceWeights, numBytesWeightGradients);
    hipMemcpy(deviceWeights, weights->filters[0].maps, numBytesWeightGradients, hipMemcpyHostToDevice);

    float* deviceGradients;
    hipMalloc((void**)&deviceGradients, numBytesGradient);
    hipMemcpy(deviceGradients, gradient.data(), numBytesGradient, hipMemcpyHostToDevice);

    int blockSize = 512;
    int gridSize = (previousLayerSize * currentLayerSize + blockSize - 1) / blockSize;

    CUDAFullyConnectedLayerBackward<<<gridSize, blockSize>>>(devicePrevGradients, deviceWeightGradients,
                                                             devicePreviousLayer,deviceWeights, deviceGradients,
                                                             previousLayerSize, currentLayerSize);

    hipMemcpy(previousGradient->inputGradients.data(), devicePrevGradients, numBytesPreviousLayer, hipMemcpyDeviceToHost);
    hipMemcpy(previousGradient->weightGradients.data(), deviceWeightGradients, numBytesWeightGradients, hipMemcpyDeviceToHost);

    hipFree(devicePrevGradients);
    hipFree(deviceWeightGradients);
    hipFree(devicePreviousLayer);
    hipFree(deviceWeights);
    hipFree(deviceGradients);

    std::memcpy(previousGradient->biasGradients.data(), gradient.data(), gradient.size() * sizeof(float));

    ClipGradient(previousGradient->inputGradients);
    ClipGradient(previousGradient->weightGradients);
    ClipGradient(previousGradient->biasGradients);

    return previousGradient;
}

void DropoutLayer(Layer *currentLayer, float dropoutRate) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> distribution(0, 1);

    for (int i = 0; i < currentLayer->width * currentLayer->height * currentLayer->depth; ++i) {
        float randomValue = distribution(gen);
        if (randomValue < dropoutRate) {
            currentLayer->maps[i] = 0;
        }
        else {
            currentLayer->maps[i] *= 1 / (1 - dropoutRate);
        }
    }
}

float MSELossFunction(const float* input, const float* predictedResult, int size) {
    float loss = 0;

    for (int i = 0; i < size; ++i) {
        float diff = input[i] - predictedResult[i];
        loss += diff * diff;
    }

    loss /= (float)size;

    return loss;
}

void ClipGradient(std::vector<float>& gradient) {
    int dataSize = (int)gradient.size();
    int numBytesDataSize = (int)(dataSize * sizeof(float));

    float* deviceData;
    hipMalloc((void**)&deviceData, numBytesDataSize);
    hipMemcpy(deviceData, gradient.data(), numBytesDataSize, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (dataSize + blockSize - 1) / blockSize;

    CUDAClipGradient<<<gridSize, blockSize>>>(deviceData, dataSize);

    hipMemcpy(gradient.data(), deviceData, numBytesDataSize, hipMemcpyDeviceToHost);
    hipFree(deviceData);
}

void UpdateNetwork(const std::vector<std::vector<Gradient*>>& gradients, std::vector<Group*>& weights, std::vector<Layer*>& biases) {
    std::vector<Gradient*> avgGradients;
    avgGradients.reserve(gradients[0].size());

    for (int i = 0; i < gradients[0].size(); ++i) {
        avgGradients.push_back(new Gradient());
        avgGradients[i]->weightGradients.resize(gradients[0][i]->weightGradients.size(), 0.0f);
        avgGradients[i]->biasGradients.resize(gradients[0][i]->biasGradients.size(), 0.0f);
    }

    for (int gradient = 0; gradient < gradients.size(); ++gradient) {
        for (int i = 0; i < gradients[gradient].size(); ++i) {
            for (int j = 0; j < gradients[gradient][i]->weightGradients.size(); ++j) {
                avgGradients[i]->weightGradients[j] += (gradients[gradient][i]->weightGradients[j] / (float)gradients.size());
            }

            for (int j = 0; j < gradients[gradient][i]->biasGradients.size(); ++j) {
                avgGradients[i]->biasGradients[j] += (gradients[gradient][i]->biasGradients[j] / (float)gradients.size());
            }
        }
    }

    UpdateWeightsAndBiases(avgGradients, weights, biases);

    for(int i = 0; i < avgGradients.size(); ++i) {
        delete avgGradients[i];
    }
}

void UpdateWeightsAndBiases(const std::vector<Gradient*>& gradients, std::vector<Group*>& weights, std::vector<Layer*>& biases) {
    AdamOptimizer* adamOptimizer = AdamOptimizer::GetInstance();

    for (int layer = 0; layer < gradients.size(); ++layer) {
        int idx = 15 - layer;
        adamOptimizer->UpdateParameters(biases[idx]->maps,  biases[idx]->width * biases[idx]->height * biases[idx]->depth,
                                        gradients[layer]->biasGradients);

        for (int i = 0; i < weights[idx]->count; ++i) {
            int weightsSize = weights[idx]->filters[i].width * weights[idx]->filters[i].height * weights[idx]->filters[i].depth;
            std::vector<float> weightGradients(weightsSize);

            std::memcpy(&weightGradients[0], &gradients[layer]->weightGradients[0] + i * weightsSize, weightsSize * sizeof(float));

            adamOptimizer->UpdateParameters(weights[idx]->filters[i].maps, weightsSize, weightGradients);
        }
    }
}
